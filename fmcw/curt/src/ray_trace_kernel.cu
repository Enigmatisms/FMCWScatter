#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "ray_trace_kernel.cuh"
#include "cuda_err_check.cuh"

// TODO: we can use dynamic allocation, but I am lazy
__device__ Vec2 all_points[MAX_PNUM];     // 1024 * 2 * 4 = 8192 bytes used
__device__ Vec2 all_normal[MAX_PNUM];     // 1024 * 2 * 4 = 8192 bytes used
__device__ ObjInfo objects[MAX_PNUM >> 2];     // 256 * 4 * 4 = 4096 bytes used (maximum allowed object number 255)
__device__ short obj_inds[MAX_PNUM];      // line segs -> obj (LUT) (material and media & AABB）(2048 bytes used)
__device__ char next_ids[MAX_PNUM];       // 1024 bytes used

void static_scene_update(
    const Vec2* const meshes, const ObjInfo* const host_objs, const short* const host_inds, 
    const char* const host_nexts, size_t line_seg_num, size_t obj_num
) {
    CUDA_CHECK_RETURN(hipMemcpy(all_points, meshes, sizeof(Vec2) * line_seg_num, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(objects, host_objs, sizeof(ObjInfo) * obj_num, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(obj_inds, host_inds, sizeof(short) * line_seg_num, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(next_ids, host_nexts, sizeof(char) * line_seg_num, hipMemcpyHostToDevice));
    // TODO: Logical check needed
    calculate_normal<<<4, get_padded_len(line_seg_num)>>>(static_cast<int>(line_seg_num));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

// block 4, thread: ceil(total_num / 4)
__global__ void calculate_normal(int line_seg_num) {
    const int pid = threadIdx.x + blockDim.x * blockIdx.x;
    if (pid < line_seg_num) {
        const int next_id = next_ids[pid];
        const int next_neg = (next_id < 0);
        const Vec2& p1 = all_points[pid];
        const Vec2& p2 = all_points[next_id * next_neg + (1 - next_neg)];
        const Vec2 dir_vec(p1.y - p2.y, p2.x - p1.x);             // perpendicular of (p2 - p1)
        all_normal[pid] = dir_vec * (1. / dir_vec.norm());  // normalized, since I didn't implement operator/
    }
    __syncthreads();
}

__forceinline__ __device__ void range_min(const float* const input, int start, int end, float& out, short& out_aux, const short* const aux = nullptr) {
    float min_depth = 9e5f;
    short min_mesh_ind = NULL_HIT;
    const bool aux_null = (aux == nullptr);
    for (int i = start; i < end; i++) {
        float local_depth = input[i];
        if (local_depth < min_depth) {
            min_depth = local_depth;
            min_mesh_ind = i * aux_null + (1 - aux_null) * aux[i];
        }
    }
    out = min_depth; 
    out_aux = min_mesh_ind; 
} 

/** 
 * @brief calculate whether a line intersects aabb 
 * input: id of an object, ray origin and ray direction
 * detailed derivation of aabb intersection should be deduced
 */
__device__ bool aabb_intersected(const Vec2& const ray_o, float dx, float dy, int obj_id) {
    const AABB& aabb = objects[obj_id].aabb;
    bool result = false, dx_valid = fabs(dx) > 2e-5f, dy_valid = fabs(dy) > 2e-5f;
    bool x_singular_valid = (ray_o.x < aabb.tr.x && ray_o.x > aabb.bl.x);     // valid condition when dx is too small
    bool y_singular_valid = (ray_o.y < aabb.tr.y && ray_o.y > aabb.bl.y);     // valid condition when dy is too small
    if (dx_valid && dy_valid) {        // there might be warp divergence (hard to resolve)
        const bool dx_pos = dx > 0, dy_pos = dy > 0;
        Vec2 act_tr(aabb.tr.x + aabb.bl.x, aabb.tr.y + aabb.bl.y);
        Vec2 act_bl(aabb.bl.x * dx_pos + aabb.tr.x * (1 - dx_pos), aabb.bl.y * dy_pos + aabb.tr.y * dy_pos);
        act_tr -= act_bl;

        const float enter_xt = (act_bl.x - ray_o.x) / dx, enter_yt = (act_bl.y - ray_o.y) / dy;
        const float exit_xt = (act_tr.x - ray_o.x) / dx, exit_yt = (act_tr.y - ray_o.y) / dy;
        const float enter_t = fmax(enter_xt, enter_yt), exit_t = fmin(exit_xt, exit_yt);
        // the following condition: or (maybe inside aabb) - and (must be outside and back-culled)
        bool back_cull = ((enter_xt < 0.f && exit_xt < 0.f) || (enter_yt < 0.f && exit_yt < 0.f));       // either pair of (in, out) being both neg, culled.
        result = (!back_cull) & (enter_t < exit_t);     // not back-culled and enter_t is smaller
    }
    result |= ((!dx_valid) & x_singular_valid);         // if x is not valid (false, ! -> true), then use x_singular_valid
    result |= ((!dy_valid) & y_singular_valid);         // if y is not valid (false, ! -> true), then use x_singular_valid
    return result;
}

// v_perp is the 90 deg rotated directional vector of current ray
__forceinline__ __device__ float ray_intersect(const Vec2& pos, const Vec2& v_perp, const Vec2& p1, const Vec2& p2) {
    const Vec2 s2e = p2 - p1;
    const Vec2 obs_v = pos - p1;
    const float D = v_perp.dot(s2e);
    float result = 1e6;
    if (fabs(D) > 5e-5) {
        float alpha = v_perp.dot(obs_v) / D;
        if (alpha < 1. && alpha > 0.) {
            float tmp = (-s2e.y * obs_v.x + s2e.x * obs_v.y) / D;
            float flag = float(tmp > 0.);
            result = tmp * flag + 1e6 * (1. - flag);
        }
    }
    return result;
}

/**
 * input : point origin (Vec2 array), ray angles: float array
 * output1 : minimum depth (float (single value, since each block represent one single ray) should be converted back to int)
 * output2 : the obj_index (of the nearest hit line seg or NULL_HIT flag)
 * @param depth is GLOBAL memory float array (for fast data copying)
 * @note this is a global function, not where the host could call. Also, AABB will not make this algo faster (only lower the power consumption)
 */
__global__ void ray_trace_cuda_kernel(
    const Vec2* const origins, const Vec2* const ray_dir, 
    RayInfo* const ray_info, short* const inds, int block_offset, int mesh_num, int aabb_num
) {
    // mem consumption: (1) mesh_num * 4 bytes (for all ranges) (2) 8 * float (min ranges, stratified) -> 32 bytes 
    // (3) 8 * short -> 4 floats -> 16 bytes (4) AABB valid bools (1 bytes * num AABB) padding
    extern __shared__ float shared_banks[];      
    bool* hit_flags = (bool*) &shared_banks[mesh_num + 12];
    const int ray_id = blockIdx.x + gridDim.x * block_offset;

    const int mesh_id = threadIdx.x + threadIdx.y * blockDim.x;
    const Vec2& ray_o = origins[ray_id];
    const Vec2& ray_d = ray_dir[ray_id];
    const float dx = ray_d.x, dy = ray_d.y;
    if (mesh_id < aabb_num) {       // first (aabb_num) threads should process AABB calculation, others remains idle
        // Bank conflict unresolvable (haven't found a very effective way)
        hit_flags[mesh_id] = aabb_intersected(ray_o, dx, dy, mesh_id);
    }
    __syncthreads();
    if (mesh_id < mesh_num) {       // for the sake of mesh (line segment division), there might be more threads than needed
        short aabb_index = obj_inds[mesh_id];
        if (hit_flags[aabb_index] == true) {        // there will be no warp divergence, since the 'else' side is NOP
            const Vec2 v_perp(-dy, dx);
            const int next_id = next_ids[mesh_id];
            const int next_id_neg = int(next_ids[mesh_id] < 0);
            const int next_pt_id = next_id * next_id_neg + 1 - next_id_neg;
            shared_banks[mesh_id] = ray_intersect(ray_o, v_perp, all_points[mesh_id], all_points[mesh_id + next_pt_id]);
            // as we use mesh indices, we don't have to store them in shared memory 
        }
    }
    __syncthreads();
    float* local_min_depths = &shared_banks[mesh_num];
    short* local_obj_inds = (short*) &shared_banks[mesh_num + 8];
    if (threadIdx.x == 0) {             // 8-thread parallel
        int max_bound = min(mesh_num, blockDim.x * (threadIdx.y + 1));
        range_min(shared_banks, blockDim.x * threadIdx.y, max_bound, local_min_depths[mesh_id], local_obj_inds[mesh_id]);
    }
    __syncthreads();
    if (mesh_id == 0) {             // only one thread attend to the final output
        RayInfo& ray = ray_info[ray_id];
        range_min(local_min_depths, 0, blockDim.y, ray.range_bound, inds[ray_id], local_obj_inds);
        ray.acc_range += ray.range_bound;
    }
    __syncthreads();
}
