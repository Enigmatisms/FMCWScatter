#include "hip/hip_runtime.h"
#include <iostream>
#include <>
#include "../include/ray_trace_kernel.cuh"
#include "../include/sampler_kernel.cuh"
#include "../include/scatter_kernel.cuh"

__device__ bool snells_law(const Vec2& inci_dir, const Vec2& norm_dir, float n1_n2_ratio, bool same_dir, float& output) {
    // if inci dir is of the same direction as the normal dir, it means that the ray is transmitting out from the media
    float sin_val = ((norm_dir.y * inci_dir.x - norm_dir.x * inci_dir.y) * n1_n2_ratio);
    bool return_flag = abs(sin_val) <= 1.0;
    if (return_flag == true) {
        float result = asinf(sin_val);
        output = (PI - result) * (1 - same_dir) + result * same_dir;
    }
    return return_flag;
}

// frensel_equation for natural light (no polarization)
__device__ float frensel_equation_natural(float n1, float n2, float cos_inc, float cos_ref) {
    float n1cos_i = n1 * cos_inc;
    float n2cos_i = n2 * cos_inc;
    float n1cos_r = n1 * cos_ref;
    float n2cos_r = n2 * cos_ref;
    float rs = (n1cos_i - n2cos_r) / (n1cos_i + n2cos_r);
    float rp = (n1cos_r - n2cos_i) / (n1cos_r + n2cos_i);
    return 0.5 * (rs * rs + rp * rp);
}

// non-deterministic branch (light interaction with surface is not decided by Material tag)
__forceinline__ __device__ void general_reflection(const Vec2& normal, const Vec2& ray_dir, hiprandState& rstate, Vec2& output, float rdist) {
    if (rdist >= 0.) {
        Vec2 reflected_dir = get_specular_dir(ray_dir, normal);
        float sampled_angle = hiprand_normal(&rstate) * fmin(0.5f, fmax(rdist, 0.f));             // 3 sigma is 1.5, which is little bit smaller than pi/2
        sampled_angle = fmaxf(fminf(sampled_angle, PI_2 - 1e-4), -PI_2 + 1e-4);             // clamp to (-pi/2 + ɛ, pi/2 - ɛ)
        Vec2 output_vec = rotate_unit_vec(reflected_dir, sampled_angle);
        const float sign = sgn(normal.dot(ray_dir));
        if (output_vec.dot(normal * sign) >= 0.) {
            output_vec = reflected_dir;
        }
        output = output_vec;      // glossy specular
    } else {
        const float sampled_angle = hiprand_uniform(&rstate) * (PI - 2e-4) - PI_2 + 1e-4;
        output = rotate_unit_vec(normal, sampled_angle);
    }
}

// Diffusive reflection light ray direction sampler
// block separation (to 8 blocks, 2048 rays).
// TODO: ray_info (rayi) and obj (ObjInfo) are not used current, but will be of use in the future (considering energy decay)
__device__ void diffusive_ref_sampler_kernel(const ObjInfo& obj, const Vec2& normal, RayInfo& rayi, Vec2* ray_d, size_t rand_offset, int ray_id) {
    hiprandState rand_state;
    hiprand_init(ray_id, 0, rand_offset + ray_id, &rand_state);
    const float sampled_angle = hiprand_uniform(&rand_state) * (PI - 2e-4) - PI_2 + 1e-4;    // we can not have exact pi/2 or -pi/2
    ray_d[ray_id] = rotate_unit_vec(normal, sampled_angle);                   // diffusive (rotate normal from -pi/2 to pi/2)
}

// Glossy object (rough specular) reflection light ray direction sampler
__device__ void glossy_ref_sampler_kernel(const ObjInfo& obj, const Vec2& normal, RayInfo& rayi, Vec2* ray_d, size_t rand_offset, int ray_id, short obj_ind) {
    hiprandState rand_state;
    hiprand_init(ray_id, 0, rand_offset + ray_id, &rand_state);
    Vec2& ray_dir = ray_d[ray_id];
    general_reflection(normal, ray_dir, rand_state, ray_dir, objects[obj_ind].rdist);       // glossy specular
}

// Mirror-like object (pure specular - Dirac BRDF) reflection light ray direction sampler
__forceinline__ __device__ void specular_ref_sampler_kernel(const ObjInfo& obj, const Vec2& normal, RayInfo& rayi, Vec2* ray_d, int ray_id) {
    ray_d[ray_id] = get_specular_dir(ray_d[ray_id], normal);   // pure specular
}

// Frensel reflection (can be reflected or refracted) - general reflection (can be diffusive, glossy or specular)
// Random number is needed here, for reflection and transmission can both happen
__device__ bool frensel_eff_sampler_kernel(const ObjInfo& object, RayInfo& rayi, Vec2& ray_d, size_t rand_offset, int ray_id, short mesh_ind) {
    const Vec2& normal = all_normal[mesh_ind];
    const float ref_index = object.ref_index;
    const float rdist = object.rdist, r_gain = object.r_gain;
    hiprandState rand_state;
    hiprand_init(ray_id, 0, rand_offset + ray_id, &rand_state);
    Vec2 refracted_dir, reflected_dir;
    general_reflection(normal, ray_d, rand_state, reflected_dir, rdist);

    float angle = 0., reflection_ratio = 1.0;
    const float cos_inc = ray_d.dot(normal), ri_sum = 1. + ref_index;         // TODO: substitude 1. to world RI 
    const bool same_dir = cos_inc > 0.;
    const float n1 = (1. - same_dir) + ref_index * same_dir;        // if same dir (out from media), n1 = ref_index, n2 = 1., else n1 = 1., n2 = ref_index
    // We do not account for transmitting from one media directly into another media
    const bool result_valid = snells_law(ray_d, normal, n1 / (ri_sum - n1), same_dir, angle);
    if (result_valid == true) {
        refracted_dir = rotate_unit_vec(normal, angle);
        reflection_ratio = frensel_equation_natural(n1, ri_sum - n1, fabs(cos_inc), fabs(cosf(angle))) * r_gain;
    }

    const bool is_reflection = hiprand_uniform(&rand_state) <= reflection_ratio;   // random choise of refracted or reflected
    ray_d = is_reflection ? reflected_dir : refracted_dir;          // warp divergence might be more efficient in this case
    return !(same_dir ^ is_reflection);         // XOR: when same_dir(1), is_ref(1) (penetrate out from medium but reflected -> is in media)
}

/// TODO: Logic failure: what if photon is not inside the object but the world (with scattering medium)?
/// The information need is not stored in objects (it should not be), but in __constant__ world
__global__ void general_interact_kernel(const short* const mesh_inds, RayInfo* const ray_info, Vec2* ray_d, size_t rand_offset) {
    const int ray_id = blockDim.x * blockIdx.x + threadIdx.x;
    const short mesh_ind = mesh_inds[ray_id];
    const short obj_ind = obj_inds[mesh_ind];
    // There is bound to be warp divergence (inevitable, or rather say, preferred)
    const ObjInfo& object = objects[obj_ind];
    const Vec2& normal = all_normal[mesh_ind];
    RayInfo& this_ray = ray_info[ray_id];
    switch (object.type) {
          case Material::DIFFUSE: {
            diffusive_ref_sampler_kernel(object, normal, this_ray, ray_d, rand_offset, ray_id); break;
        } case Material::GLOSSY: {
            glossy_ref_sampler_kernel(object, normal, this_ray, ray_d, rand_offset, ray_id, obj_ind); break;
        } case Material::SPECULAR: {
            specular_ref_sampler_kernel(object, normal, this_ray, ray_d, ray_id); break;
        } case Material::REFRACTIVE: {
            // TODO: more things should be accounted for: world refraction index, transimitting from media 1 to media 2
            frensel_eff_sampler_kernel(object, this_ray, ray_d[ray_id], rand_offset, ray_id, mesh_ind); break;
        } case Material::SCAT_ISO: {
            scattering_interaction(object, this_ray, ray_d[ray_id], isotropic_phase, ray_info[ray_id].is_in_media, ray_id, mesh_ind, rand_offset); break;
        } case Material::SCAT_HG: {
            scattering_interaction(object, this_ray, ray_d[ray_id], henyey_greenstein_phase, ray_info[ray_id].is_in_media, ray_id, mesh_ind, rand_offset); break;
        } case Material::SCAT_RAYLEIGH: {
            scattering_interaction(object, this_ray, ray_d[ray_id], rayleigh_phase, ray_info[ray_id].is_in_media, ray_id, mesh_ind, rand_offset); break;
        } default: {
            break;
        }
    }
    // Massive warp divergence and serialization
    __syncthreads();
}
