#include "ray_trace_host.cuh"
#include "sampler_kernel.cuh"
#include "cuda_err_check.cuh"

#define BLOCK_PER_STREAM 32

// CPU end: the last commit when range_ptr and mesh_inds are available is c4815846c
PathTracer::PathTracer(size_t ray_num):
    ray_os(nullptr, get_deletor<Vec2>()), intersects(nullptr, get_deletor<Vec2>()),
    ray_d(nullptr, get_deletor<Vec2>()), ray_num(ray_num)
{
    // actually, ranges will be summed (in RayInfo)
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_ray_os, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_ray_d, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_intersects, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_ray_info, ray_num * sizeof(RayInfo)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_mesh_inds, ray_num * sizeof(short)));
    // pinned memory allocation, let's do some math: suppose there are 2048 rays
    // 2048 * 4 * (2 + 2 + 1 + 1 + 1) = 56 KB (only 56KB pinned memory is allocated!)
    // ray_os is definitely needed, since we wish to change the position of the FMCW scanner in the future
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &ray_os_ptr, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &ray_d_ptr, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &intersect_ptr, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    ray_os.reset(ray_os_ptr);
    ray_d.reset(ray_d_ptr);
    intersects.reset(intersect_ptr);
}

PathTracer::~PathTracer() {
    CUDA_CHECK_RETURN(hipFree(cu_ray_os));
    CUDA_CHECK_RETURN(hipFree(cu_ray_d));
    CUDA_CHECK_RETURN(hipFree(cu_intersects));
    CUDA_CHECK_RETURN(hipFree(cu_mesh_inds));
    CUDA_CHECK_RETURN(hipFree(cu_ray_info));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

void PathTracer::next_intersections(bool host_update, int mesh_num, int aabb_num) {
    if (host_update == true) {          // if not, it means that we are doing path tracing (otherwise it is the first path tracing given a new pose)
        CUDA_CHECK_RETURN(hipMemcpy(cu_ray_os, ray_os_ptr, ray_num * sizeof(Vec2), hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpy(cu_ray_d, ray_d_ptr, ray_num * sizeof(Vec2), hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
    }
    
    hipStream_t streams[8];
    for (short i = 0; i < 8; i++)
        hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
    const int cascade_num = ray_num / BLOCK_PER_STREAM;
    size_t shared_mem_size = (ray_num << 2) + 48 + get_padded_len(aabb_num);
    size_t threads_along_x = get_padded_len(mesh_num);
    for (int i = 0; i < cascade_num; i++) {
        ray_trace_cuda_kernel<<<BLOCK_PER_STREAM, dim3(threads_along_x, 8), shared_mem_size, streams[i % 8]>>>(
            cu_ray_os, cu_ray_d, cu_ray_info, cu_mesh_inds, i, mesh_num, aabb_num
        );
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    for (int i = 0; i < 8; i++)
        hipStreamDestroy(streams[i]);
    CUDA_CHECK_RETURN(hipMemcpy(intersect_ptr, cu_intersects, ray_num * sizeof(Vec2), hipMemcpyDeviceToHost));
    // TODO: Do we really need range output and mesh_inds output (to CPU end?)
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

void PathTracer::sample_outgoing_rays() {
    static size_t random_offset = 0;
    // update the intersections (ray origin updates from original starting point to intersection points) 
    CUDA_CHECK_RETURN(hipMemcpy(cu_ray_os, cu_intersects, ray_num * sizeof(Vec2), hipMemcpyDeviceToDevice));  // assume this copy operation won't emit exception
    
    // within this function, there is nothing to be fetched multiple times, therefore shared memory is not needed.
    // update the ray direction, in order to get next intersection
    non_scattering_interact_kernel<<< 8, (ray_num >> 3) >>>(cu_mesh_inds, cu_ray_info, cu_ray_d, random_offset);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    random_offset += 1;
}
