#include <algorithm>
#include "ray_trace_host.cuh"
#include "sampler_kernel.cuh"
#include "cuda_err_check.cuh"

#define BLOCK_PER_STREAM 32

PathTracer::PathTracer():
    ray_os(nullptr, get_deletor<Vec2>()), intersects(nullptr, get_deletor<Vec2>()),
    ray_d(nullptr, get_deletor<Vec2>()), ray_num(0) {}

// CPU end: the last commit when range_ptr and mesh_inds are available is c4815846c
PathTracer::PathTracer(size_t ray_num):
    ray_os(nullptr, get_deletor<Vec2>()), intersects(nullptr, get_deletor<Vec2>()),
    ray_d(nullptr, get_deletor<Vec2>()), ray_num(ray_num)
{
    setup(ray_num);
}

void PathTracer::setup(size_t ray_num) {
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_ray_os, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_ray_d, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_intersects, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_ray_info, ray_num * sizeof(RayInfo)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_mesh_inds, ray_num * sizeof(short)));
    // pinned memory allocation, let's do some math: suppose there are 2048 rays
    // 2048 * 4 * (2 + 2 + 1 + 1 + 1) = 56 KB (only 56KB pinned memory is allocated!)
    // ray_os is definitely needed, since we wish to change the position of the FMCW scanner in the future
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &ray_os_ptr, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &ray_d_ptr, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &intersect_ptr, ray_num * sizeof(Vec2)));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    ray_os.reset(ray_os_ptr);
    ray_d.reset(ray_d_ptr);
    intersects.reset(intersect_ptr);
}

PathTracer::~PathTracer() {
    CUDA_CHECK_RETURN(hipFree(cu_ray_os));
    CUDA_CHECK_RETURN(hipFree(cu_ray_d));
    CUDA_CHECK_RETURN(hipFree(cu_intersects));
    CUDA_CHECK_RETURN(hipFree(cu_mesh_inds));
    CUDA_CHECK_RETURN(hipFree(cu_ray_info));

    CUDA_CHECK_RETURN(hipFree(cu_ray_info));
    CUDA_CHECK_RETURN(hipFree(cu_ray_info));
    CUDA_CHECK_RETURN(hipFree(cu_ray_info));

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

// How to call the function from the host side? We should be able to access any data we need
void PathTracer::next_intersections(bool host_update, int mesh_num, int aabb_num) {
    if (host_update == true) {          // if not, it means that we are doing path tracing (otherwise it is the first path tracing given a new pose)
        CUDA_CHECK_RETURN(hipMemcpy(cu_ray_os, ray_os_ptr, ray_num * sizeof(Vec2), hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpy(cu_ray_d, ray_d_ptr, ray_num * sizeof(Vec2), hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
    }
    
    hipStream_t streams[8];
    for (short i = 0; i < 8; i++)
        hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
    const int cascade_num = ray_num / BLOCK_PER_STREAM;
    size_t shared_mem_size = (ray_num << 2) + 48 + pad_bytes(aabb_num);
    size_t threads_along_x = get_padded_len(mesh_num, 8.);
    for (int i = 0; i < cascade_num; i++) {
        ray_trace_cuda_kernel<<<BLOCK_PER_STREAM, dim3(threads_along_x, 8), shared_mem_size, streams[i % 8]>>>(
            cu_ray_os, cu_ray_d, cu_ray_info, cu_mesh_inds, i, mesh_num, aabb_num
        );
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    for (int i = 0; i < 8; i++)
        hipStreamDestroy(streams[i]);
    CUDA_CHECK_RETURN(hipMemcpy(intersect_ptr, cu_intersects, ray_num * sizeof(Vec2), hipMemcpyDeviceToHost));
    // TODO: Do we really need range output and mesh_inds output (to CPU end?)
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

void PathTracer::sample_outgoing_rays() {
    static size_t random_offset = 0;
    
    // within this function, there is nothing to be fetched multiple times, therefore shared memory is not needed.
    // update the ray direction, in order to get next intersection
    non_scattering_interact_kernel<<< 8, (ray_num >> 3) >>>(cu_mesh_inds, cu_ray_d, random_offset);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // update the intersections (ray origin updates from original starting point to intersection points) 
    CUDA_CHECK_RETURN(hipMemcpy(cu_ray_os, cu_intersects, ray_num * sizeof(Vec2), hipMemcpyDeviceToDevice));  // assume this copy operation won't emit exception
    random_offset += 1;
}


extern "C" {
    // Whether this can reside here 
    PathTracer path_tracer;

    void setup_path_tracer(int ray_num) {
        path_tracer.setup(static_cast<size_t>(ray_num));
    }

    // Get intersection of the light rays and update the ray directions and ray origins
    void get_intersections_update(Vec2* const intersections, bool host_update, int mesh_num, int aabb_num) {
        path_tracer.next_intersections(host_update, mesh_num, aabb_num);
        std::copy_n(path_tracer.intersects.get(), path_tracer.get_ray_num(), intersections);
        path_tracer.sample_outgoing_rays();
    }
}
